#include "hip/hip_runtime.h"
#include "server.hpp"
#define USED_SLOT 128
#define VALID_SLOT 124



PirServer::PirServer(const EncryptionParameters &enc_params,
                     const PirParams &pir_params, CAHELGPUContext &context, CAHELGPUBatchEncoder &encode)
    : enc_params(enc_params), pir_params(pir_params) {
  //context_ = make_shared<CAHELGPUContext>(enc_params, true,cahel::sec_level_type::tc128);
  //evaluator_ = make_unique<Evaluator>(context_);
  //encoder_ = make_unique<CAHELGPUBatchEncoder>(context);
  //encoder_ = encode;
  single_map(context,encode);
}

void PirServer::set_database(std::unique_ptr<Database> &&db) {
  db_ = move(db);
}


void PirServer::set_database(CAHELGPUContext &context_,CAHELGPUBatchEncoder &encoder_,const std::unique_ptr<const int8_t[]> &bytes, std::int64_t ele_num, std::int64_t ele_size) {
    int32_t logt = floor(log2(enc_params.plain_modulus().value()));
    int32_t N = enc_params.poly_modulus_degree();
    auto result = make_unique<vector<CAHELGPUPlaintext>>();
    int num_CAHELGPUPlaintext = ele_num / N * ele_size / 2;
    vector<vector<int64_t>> coefficients = vector<vector<int64_t>>(num_CAHELGPUPlaintext, vector<int64_t>(N,0));//just see one coefficient contain two bytes
    //vector<int64_t> tag(2020,0);//every byte from 0 - 20
    cout<<"Elements num is "<<ele_num<<endl;
    int offset = 0;
    //guarantee the minum size of CAHELGPUPlaintext
    for(int64_t j = 0;j< 2 * num_CAHELGPUPlaintext/ele_size;j++){ //which slot  column
            for(int64_t i = 0;i< N;i++)  //which slot row
            { 
                for(int64_t k = 0;k<ele_size/2;k++)
                {   
                    int64_t coeff = bytes_to_coeffs(logt,bytes.get()+offset,2);
                    offset += 2;
                    coefficients[k + j* ele_size/2][i] = coeff;
                }
                
            }
    }
    cout<<"database palintexts size is "<<coefficients.size()<<endl;
   //now we can encode these coefficients into CAHELGPUPlaintexts
    for(int64_t i = 0;i< coefficients.size();i++)
    {
        CAHELGPUPlaintext p(context_);
        encoder_.encode(context_,coefficients[i],p);
        result->push_back(move(p));
    }
    //db_ = make_unique<Database>(std::move(result));
    set_database(std::move(result));
}


void PirServer::process_database() {
    
}
void PirServer::single_map(CAHELGPUContext &context_,CAHELGPUBatchEncoder &encoder_)
{
    /* this i represent the data-item,for example , data-item  contains four bytes,
      the field is two bytes,and every bytes is from 00 - 20,so we can represent the two bytes as first byte * 100 and last byte *1,
      and the range is from 0000(0) ~ 2020*/
      cout<<"Map the keywords"<<endl;
    int64_t N = enc_params.poly_modulus_degree();
    vector<vector<int64_t>> coeffcients;
    for(int64_t i = 0;i<=pir_params.ele_num;i++)
    {
        /* generate the unique constant weight map
         and we can map the dataitem to only one codeword 
        */
        vector<int64_t> coeff = constant_weight_map(i,VALID_SLOT,pir_params.k);
        // for(int j = 0;j<coeff.size();j++)
        // {
        //     cout<<coeff[j]<<" ";
        // }
        // cout<<endl;
        coeffcients.push_back(coeff);
    }
    int64_t column = pir_params.ele_num / N;
    vector<vector<int64_t>> result= vector<vector<int64_t>>(column * VALID_SLOT, vector<int64_t>(N,0));
    for(int i = 0;i<column;i++)
    {
        for(int j = 0;j<N;j++)
        {
            for(int k = 0;k<VALID_SLOT;k++){
                result[i * VALID_SLOT + k][j] = coeffcients[j + i*N][k];
            }
        }
    }
    for(int64_t i = 0;i<result.size();i++)
    {
        result[i][N-1] = 1;// prevent all 0
        //cout<<"pt generated"<<endl;
        CAHELGPUPlaintext pt(context_);
        encoder_.encode(context_,result[i],pt);
        //cout<<"pt mapped"<<endl;
        map_[i] = move(pt);
    }
    
}

int PirServer::serialize_reply(CAHELGPUContext &context_,PirReply &reply,std::stringstream &stream)
{
    int output_size = 0;
    mod_switch_to_inplace(context_,reply,context_.cpu_context_->last_parms_id());
    //output_size += reply.save(stream);
    reply.save(stream);
    return output_size;
}
PirReply PirServer::generate_reply(CAHELGPUContext &context_,CAHELGPUBatchEncoder &encoder_,PirQuery query,stringstream &stream,CAHELGPUGaloisKey &gal_keys,CAHELGPURelinKey &rel_keys){
    CAHELGPUGaloisKey rotate_galois_(context_);
    rotate_galois_.load(context_,stream);
    int64_t N = enc_params.poly_modulus_degree();
    vector<CAHELGPUPlaintext> *cur = db_.get();
    //vector<CAHELGPUCiphertext> expanded_query = expand_query(query,USED_SLOT);
    cout<<"Server : expanded over"<<endl;
    int column = (*cur).size()/(pir_params.ele_size/2);
    cout<<"Server: expanded query multiply the keyword"<<endl;
    vector<CAHELGPUCiphertext> keyword_CAHELGPUCiphertexts;
    for(int j = 0;j<column;j++){
        for(int i = 0;i<VALID_SLOT;i++)
        {
            CAHELGPUCiphertext tag(context_);
            multiply_plain(context_,query[i],map_[i + j * VALID_SLOT],tag);
            keyword_CAHELGPUCiphertexts.push_back(tag);
        }
    }
    //add the keyword slot based on block encoding
    cout<<"Server : add the keyword slot based on block encoding"<<endl;
    vector<vector<CAHELGPUCiphertext>> inter;
    
    for(int k = 0;k< column;k++){
        vector<CAHELGPUCiphertext> inter_cipher;
        for(int i = 0;i<pir_params.k;i++)
        {
            CAHELGPUCiphertext temp = keyword_CAHELGPUCiphertexts[i*pir_params.block_num + k * pir_params.m]; //pir_params.m is the keyword CAHELGPUCiphertext in every column
            for (int j = 1; j < pir_params.block_num; j++)
            {
                /* code */
                add_inplace(context_,temp,keyword_CAHELGPUCiphertexts[i * pir_params.block_num + j + k * pir_params.m]);
                
            }
            inter_cipher.push_back(temp);
        }
        inter.push_back(inter_cipher);
    }
    
    
    cout<<"Server: start find the desire slot in keyword codeword"<<endl;
    vector<vector<CAHELGPUCiphertext>> result_db;
    for(int i = 0;i<column;i++)
    {
        CAHELGPUCiphertext result = inter[i][0];
        multiply_many(context_,inter[i],rel_keys,result);
        //CAHELGPUCiphertext temp = inter[i][0];
        cout<<"multiply many over"<<endl;
        cout<<"all will mul_plain times are "<<pir_params.ele_size/2<<endl;
        vector<CAHELGPUCiphertext> temp_db;
        //temp_db.resize(pir_params.ele_size/2);
        for(int j = 0;j<pir_params.ele_size/2;j++)
        {
            CAHELGPUCiphertext vec_result(context_);
            multiply_plain(context_,result,(*cur)[i*pir_params.ele_size/2 + j],vec_result); //multiply the database
            temp_db.push_back(vec_result);
        }
        cout<<"The "<<i<<"-th column over!"<<endl;
        result_db.push_back(temp_db);
    }
    vector<int64_t> vec(N,1);
    vec[N-1] = 0;
    CAHELGPUPlaintext pt(context_);
    encoder_.encode(context_,vec,pt);
    cout<<"next will add database entry !!!"<<endl;
    //database entry add 
    for(int i = 0;i<result_db[0].size();i++)
    {
        for(int j = 1;j<column;j++)
        {
            add_inplace(context_,result_db[0][i],result_db[j][i]);
        }
        multiply_plain_inplace(context_,result_db[0][i],pt);
    }
    for(int i = 0;i<result_db[0].size();i++)
    {
        //cout<<"rotate and add"<<endl;
        //rotate_rows_inplace(context_,result_db[0][i],i,rotate_galois_);
        rotate_rows_inplace(context_,result_db[0][i],i,gal_keys);
    }
    CAHELGPUCiphertext result_in_all = result_db[0][0];
    for(int i = 1;i<result_db[0].size();i++)
    {
        add_inplace(context_,result_in_all,result_db[0][i]);
    }
    return result_in_all;
    
}

/*
inline vector<CAHELGPUCiphertext> PirServer::expand_query(const CAHELGPUCiphertext &encrypted,int32_t need_slot)
{
    CAHELGPUGaloisKeys &galkey = galoisKeys_;

    // Assume that m is a power of 2. If not, round it to the next power of 2.
    int32_t logm = ceil(log2(need_slot));
    CAHELGPUPlaintext two("2");

    vector<int> galois_elts;
    auto n = enc_params.poly_modulus_degree();
    if (logm > ceil(log2(n))) {
        throw logic_error("m > n is not allowed.");
    }
    for (int i = 0; i < ceil(log2(n)); i++) {
        galois_elts.push_back((n + exponentiate_int(2, i)) /
                            exponentiate_int(2, i));
    }

    vector<CAHELGPUCiphertext> temp;
    temp.push_back(encrypted);
    CAHELGPUCiphertext tempctxt;
    //rotated / shifted /rotatedshifted ???
    CAHELGPUCiphertext tempctxt_rotated;
    CAHELGPUCiphertext tempctxt_shifted;
    CAHELGPUCiphertext tempctxt_rotatedshifted;
    //section 3.3 in Sealpir's paper,figure 3
    for (int32_t i = 0; i < logm - 1; i++) {
        vector<CAHELGPUCiphertext> newtemp(temp.size() << 1);
        // temp[a] = (j0 = a (mod 2**i) ? ) : Enc(x^{j0 - a}) else Enc(0).  With
        // some scaling....
        int index_raw = (n << 1) - (1 << i);
        int index = (index_raw * galois_elts[i]) % (n << 1);

        for (int32_t a = 0; a < temp.size(); a++) {

        evaluator_->apply_galois(temp[a], galois_elts[i], galkey,
                                tempctxt_rotated);

        // cout << "rotate " <<
        // client.decryptor_->invariant_noise_budget(tempctxt_rotated) << ", ";

        evaluator_->add(temp[a], tempctxt_rotated, newtemp[a]);
        multiply_power_of_X(temp[a], tempctxt_shifted, index_raw);

        // cout << "mul by x^pow: " <<
        // client.decryptor_->invariant_noise_budget(tempctxt_shifted) << ", ";
        
        multiply_power_of_X(tempctxt_rotated, tempctxt_rotatedshifted, index);

        // cout << "mul by x^pow: " <<
        // client.decryptor_->invariant_noise_budget(tempctxt_rotatedshifted) <<
        // ", ";

        // Enc(2^i x^j) if j = 0 (mod 2**i).
        evaluator_->add(tempctxt_shifted, tempctxt_rotatedshifted,
                        newtemp[a + temp.size()]);
        }
        temp = newtemp;
        /*
        cout << "end: ";
        for (int h = 0; h < temp.size();h++){
            cout << client.decryptor_->invariant_noise_budget(temp[h]) << ", ";
        }
        cout << endl;
        
    }
    // Last step of the loop
    vector<CAHELGPUCiphertext> newtemp(temp.size() << 1);
    int index_raw = (n << 1) - (1 << (logm - 1));
    int index = (index_raw * galois_elts[logm - 1]) % (n << 1);
    for (int32_t a = 0; a < temp.size(); a++) {
        if (a >= (need_slot - (1 << (logm - 1)))) { // corner case.
        evaluator_->multiply_plain(temp[a], two,
                                    newtemp[a]); // plain multiplication by 2.
        // cout << client.decryptor_->invariant_noise_budget(newtemp[a]) << ", ";
        } else {
        evaluator_->apply_galois(temp[a], galois_elts[logm - 1], galkey,
                                tempctxt_rotated);
        evaluator_->add(temp[a], tempctxt_rotated, newtemp[a]);
        multiply_power_of_X(temp[a], tempctxt_shifted, index_raw);
        multiply_power_of_X(tempctxt_rotated, tempctxt_rotatedshifted, index);
        evaluator_->add(tempctxt_shifted, tempctxt_rotatedshifted,
                        newtemp[a + temp.size()]);
        }
    }

    vector<CAHELGPUCiphertext>::const_iterator first = newtemp.begin();
    vector<CAHELGPUCiphertext>::const_iterator last = newtemp.begin() + need_slot;
    vector<CAHELGPUCiphertext> newVec(first, last);

    return newVec;
}
*/

// PirQuery PirServer::deserialize_query(stringstream &stream)
// {
//   PirQuery q;
//   /*
//   int32_t ctx = ceil((pir_params_.num_ofCAHELGPUPlaintexts + 0.0)/enc_params_.poly_modulus_degree());
//   vector<CAHELGPUCiphertext> cs;
//   for(int32_t i=0; i<ctx; i++)
//   {
//     CAHELGPUCiphertext c;
//     c.load(context_,stream);
//     cs.push_back(c);
//   }
//   q = cs;
//   */
//   vector<CAHELGPUCiphertext> c ;
//   c.load(context_,stream);
//   q=c;
//   return q;
// }

void PirServer::deserialize_relinkkeys(CAHELGPUContext &context_,stringstream &stream)
{
  this->relin_keys.load(context_,stream);
}
/*
inline void PirServer::multiply_power_of_X(const CAHELGPUCiphertext &encrypted, CAHELGPUCiphertext &destination, int32_t index){
    auto coeff_mod_count = enc_params.coeff_modulus().size() - 1;
    auto coeff_count = enc_params.poly_modulus_degree();
    auto encrypted_count = encrypted.size();

    // cout << "coeff mod count for power of X = " << coeff_mod_count << endl;
    // cout << "coeff count for power of X = " << coeff_count << endl;

    // First copy over.
    destination = encrypted;

    // Prepare for destination
    // Multiply X^index for each CAHELGPUCiphertext polynomial
    for (int i = 0; i < encrypted_count; i++) {
        for (int j = 0; j < coeff_mod_count; j++) {
        negacyclic_shift_poly_coeffmod(encrypted.data(i) + (j * coeff_count),
                                        coeff_count, index,
                                        enc_params.coeff_modulus()[j],
                                        destination.data(i) + (j * coeff_count));
        }
    }
}
*/
/*CAHELGPUCiphertext PirServer::equality_operator(CAHELGPUCiphertext &ct, int32_t k){
   // cout<<"equality_operator k is"<<k<<endl;
    int64_t m = 1;
    for(int32_t i=k; i >0;i--)
    {
        m *= i;
    }
    int64_t inverse = 0;
    inverse = invert_mod(m, enc_params.plain_modulus());
    vector<CAHELGPUCiphertext> cts;
    for(int64_t i = 0; i < k;i++)
    {
        CAHELGPUPlaintext pt(int_to_hex_string(&i,std::size_t(1)));
        CAHELGPUCiphertext ct1;
        evaluator_->sub_plain(ct, pt, ct1);
        cts.push_back(ct1);
    }
    CAHELGPUCiphertext result;
    evaluator_->multiply_many(cts, relin_keys,result);
  //  cout<<"multiply many finish"<<endl;
    CAHELGPUPlaintext pt1(int_to_hex_string(&inverse,std::size_t(1)));
    evaluator_->multiply_plain_inplace(result, pt1);
    return result;
}*/

   vector<CAHELGPUPlaintext> PirServer::get_slot_pt(CAHELGPUContext &context_,CAHELGPUBatchEncoder &encoder_)
    {
        vector<CAHELGPUPlaintext> pts;
        for(int i =0;i<2020;i++)
        {
            vector<int64_t> vec(enc_params.poly_modulus_degree(),0);
            vec[i] = 1;
            CAHELGPUPlaintext pt(context_);
            encoder_.encode(context_,vec, pt);
            pts.push_back(pt);
        }
        return pts;
    }

   /* void PirServer::deserialized_galois(stringstream &stream)
    {
        rotate_galois_.load(stream);
    }*/
