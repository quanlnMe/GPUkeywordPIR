#include "hip/hip_runtime.h"
#include "pir.hpp"
#define VALID_SLOT 124
using namespace cahel;
using namespace cahel::util;


void gen_encrypt_params(int32_t N, std::int32_t logt, EncryptionParameters &enc_params){
  enc_params.set_poly_modulus_degree(N);
  enc_params.set_coeff_modulus(CoeffModulus::BFVDefault(N));
  enc_params.set_plain_modulus(PlainModulus::Batching(N, logt + 1));
  //const vector<int> galois_steps = {1,2};
  //auto galois_elts = get_elts_from_steps(galois_steps, N);
  //enc_params.set_galois_elts(galois_elts);

  // the +1 above ensures we get logt bits for each plaintext coefficient.
  // Otherwise the coefficient modulus t will be logt bits, but only floor(t) =
  // logt-1 (whp) will be usable (since we need to ensure that all data in the
  // coefficient is < t).
}

void gen_pir_params(int64_t number_of_itmes,int64_t size_per_item,PirParams &pirparams){
  pirparams.ele_num = number_of_itmes;
  pirparams.ele_size = size_per_item;
  pirparams.k = 4;
  pirparams.m = VALID_SLOT;
  pirparams.block_num = pirparams.m / pirparams.k;

}

std::int64_t byte_num_per_coefficient(std::int32_t logt)
{
    return floor(logt / 8);
}

std::int64_t bytes_to_coeffs(int32_t limit,const int8_t *bytes, int64_t size)
{
  //cout<<"PIR:start to convert bytes to coefficients"<<endl;
    int64_t ele_size = byte_num_per_coefficient(limit);
    int64_t coeffs;
    for(int64_t i = 0; i < size; i++)
    {
        coeffs <<= 8;
        coeffs |= bytes[i];
    }
    //cout<<"One converted"<<endl;
    return coeffs;
}

void coeffs_to_bytes(std::vector<int64_t> coeffs, int64_t size,int64_t ele_size, int8_t *output)
{
    //output = new uint8_t[size * ele_size];
    for(int64_t i = 0; i < size; i++)
    {
        for(int64_t j = 0; j < 2; j++) //every coefficient contains two bytes
        {
            output[i * 2 + j] = coeffs[i] >> (8 * j);
        }
    }
}

uint64_t invert_mod(uint64_t m, const cahel::Modulus &mod) {
  if (mod.uint64_count() > 1) {
    cout << "Mod too big to invert";
  }
  uint64_t inverse = 0;
  if (!cahel::util::try_invert_uint_mod(m, mod.value(), inverse)) {
    cout << "Could not invert value";
  }
  return inverse;
}
