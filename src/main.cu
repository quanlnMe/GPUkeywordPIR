#include "hip/hip_runtime.h"
#include "pir.hpp"
#include "client.hpp"
#include <iostream>
#include "server.hpp"
#include <random>
#include "CAHEL/cahel.h"
#include <chrono>

using namespace std;
using namespace std::chrono;

int main(int argc, char *argv[]){
    int64_t number_of_itmes = 262144;   
    int32_t logt = 20;
    int64_t size_per_item = 1024;
    int32_t N = 8192;

    EncryptionParameters enc_params(scheme_type::bfv);
    PirParams pir_params;
    cout<<"Main : Generating SEAL PArameters"<<endl;
    gen_encrypt_params(N, logt, enc_params);
    gen_pir_params(number_of_itmes,size_per_item,pir_params);
    auto context_ = CAHELGPUContext(enc_params,true,cahel::sec_level_type::tc128);
    CAHELGPUSecretKey sk(enc_params);
    sk.gen_secretkey(context_);
    CAHELGPURelinKey rlk(context_);
    sk.gen_relinkey(context_,rlk);
    CAHELGPUGaloisKey glk(context_);
    sk.create_galois_keys(context_,glk);
    CAHELGPUBatchEncoder encoder_(context_);

    PirClient pir_client(enc_params,pir_params,context_, encoder_);
    cout<<"Main : Generating galois_keys"<<endl;
    stringstream gal_stream;
    //CAHELGPUGaloisKey rotate_galois = pir_client.generate_rotate_galois();
    pir_client.generate_serialized_rotate_galois(context_,gal_stream);
    cout<<"Initializing server"<<endl;
    PirServer pir_server(enc_params,pir_params,context_, encoder_);
    //pir_server.set_rotate_galois(rotate_galois);
    //pir_server.deserialized_galois(stream);
    cout<<"Initializing database"<<endl;
    cout<<"Main: Creating the database with random data "<<endl;

    auto db(make_unique<int8_t[]>(number_of_itmes*size_per_item));

    random_device rd;
    for(int64_t i = 0;i<number_of_itmes;i++)
    {
        for(int64_t j = 0;j<size_per_item;j++)
        {
            int8_t val = rd() % 255;
            db.get()[i*size_per_item+j] = val;
        }
    }
    cout<<"Main: Starting to process the database"<<endl;
    pir_server.process_database();
    pir_server.set_database(context_,encoder_,move(db),number_of_itmes,size_per_item);
    cout<<"database generated!"<<endl;

    cout<<"Generate random query keyword"<<endl;
    // int64_t field1 = (int64_t)db.get()[rd() % (number_of_itmes * size_per_item)];
    // cout<<field1<<endl;
    // int64_t field2 = (int64_t)db.get()[rd() % (number_of_itmes * size_per_item)];
    // int64_t field = field1 *100+ field2;
    int64_t field = rd() % number_of_itmes;
    cout<<"Main : You want to fuzzy query all items containing "<<field<<endl;

    auto time_query_s = high_resolution_clock::now();
    PirQuery query = pir_client.generate_query(context_,encoder_,field,sk);
    auto time_query_e = high_resolution_clock::now();
    auto time_query = duration_cast<microseconds>(time_query_e - time_query_s);
    cout<<"query generated!"<<endl;
    //cout<<"Main : Query time is "<<time_query.count()<<endl;

    stringstream client_stream;
    stringstream server_stream;
    stringstream relink_stream;

    int relink_size = pir_client.generate_serialized_relinkKey(context_,relink_stream);
    cout<<"Main : Relink size is "<<relink_size<<endl;
    pir_server.deserialize_relinkkeys(context_,relink_stream);

    auto time_s_query_s = high_resolution_clock::now();
    int query_size = pir_client.generate_serialized_query(context_,encoder_,field,client_stream);
    auto time_s_query_e = high_resolution_clock::now();
    auto time_s_query = duration_cast<microseconds>(time_s_query_e - time_s_query_s);
    cout<<"query serialized!"<<endl;
    //cout<<"Main : Serialization time is "<<time_s_query.count()<<endl;

    auto time_desierial_s = high_resolution_clock::now();
    //PirQuery query2 = pir_server.deserialize_query(client_stream);
    auto time_desierial_e = high_resolution_clock::now();
    auto time_desierial = duration_cast<microseconds>(time_desierial_e - time_desierial_s);
    cout<<"Query deserialized!"<<endl;
    //cout<<"Main : Deserialization time is "<<time_desierial.count()<<endl;
   

   auto time_server_s = high_resolution_clock::now();
   PirReply reply = pir_server.generate_reply(context_,encoder_,query,gal_stream,glk,rlk);
   auto time_server_e = high_resolution_clock::now();
   auto time_server = duration_cast<microseconds>(time_server_e - time_server_s);
   cout<<"Reply generated!"<<endl;
   //cout<<"Main : Server time is "<<time_server.count()<<endl;

   int reply_size = pir_server.serialize_reply(context_,reply,server_stream);

    vector<int8_t> elems = pir_client.decode_reply(context_,encoder_,reply,sk);
    for(int64_t i = 0;i<elems.size();i++)
    {
        cout<<(int)elems[i]<<" ";
    }
    cout<<endl;

   cout<<"Main: PIR result conrrect!"<<endl;
   cout<<"------------------GPU :-------------------"<<endl;
   cout<<"Main: PIRClient query time is "<<ceil(time_query.count()/1000)<<endl;
   cout<<"Main: PIRClient serialization time is "<<ceil(time_s_query.count()/1000)<<endl;
   cout<<"Main: PIRClient deserialization time is "<<std::ceil(time_desierial.count()/1000)<<endl;
   cout<<"Main: PIRServer reply time is "<<ceil(time_server.count()/1000)<<endl;


   return 0;



}
